#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cv.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui.hpp>
#include <highgui.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "cvui.h" // For interfaces

using namespace std;
using namespace cv;

# define PI 3.1416
# define WINDOW_NAME "ImageEditor"

#define CANVAS_WIDTH  960
#define CANVAS_HEIGHT 960

# define MENU_TRANSFORM         1
# define MENU_EQUALIZAR         2
# define MENU_CONVOLUCION       3
# define MENU_FOURIER                4
# define MENU_PATTERN_SEARCH    5
# define MENU_MORPH             6

int MENU        = 1;
int COLUMN_1    = 10;
int COLUMN_2    = 200;
int COLUMN_3    = 850;
int ROW_HEIGHT  = 35;
int ROW_ACTUAL  = 0;

dim3 blockDims;
dim3 gridDims;
hipEvent_t start, stop;
unsigned char *gpu_image_output, *gpu_image_input, *gpu_image_temp, *gpu_image_aux;
unsigned char *cpu_image_output, *cpu_image_input, *cpu_image_aux;
Mat image_input, image_canvas, image_output, image_aux, image_temp;
bool pattern_loaded = false;
bool image_loaded = false;
float milliseconds = 0.0;
string file_path = "";

struct MatPixel {
    uchar b;
    uchar g;
    uchar r;
};

__global__ void kernel_Gaussiano(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    if (x < width - 3 && x > 3 && y < height - 3 && y > 3) {
        int p11 = offset - 2 * width - 2;
        int p12 = offset - 2 * width - 1;
        int p13 = offset - 2 * width;
        int p14 = offset - 2 * width + 1;
        int p15 = offset - 2 * width + 2;

        int p21 = offset - width - 2;
        int p22 = offset - width - 1;
        int p23 = offset - width;
        int p24 = offset - width + 1;
        int p25 = offset - width + 2;

        int p31 = offset - 2;
        int p32 = offset - 1;
        int p33 = offset;
        int p34 = offset + 1;
        int p35 = offset + 2;

        int p41 = offset + width - 2;
        int p42 = offset + width - 1;
        int p43 = offset + width;
        int p44 = offset + width + 1;
        int p45 = offset + width + 2;

        int p51 = offset + 2 * width - 2;
        int p52 = offset + 2 * width - 1;
        int p53 = offset + 2 * width;
        int p54 = offset + 2 * width + 1;
        int p55 = offset + 2 * width + 2;

        //if ( offset < height * width) {
        output_image[offset * 3] = (
                                       2 * input_image[p11 * 3] +  4 * input_image[p12 * 3]  +  5 * input_image[p13 * 3] +  4 * input_image[p14 * 3] + 2 * input_image[p15 * 3] +
                                       4 * input_image[p21 * 3] +  9 * input_image[p22 * 3]  + 12 * input_image[p23 * 3] +  9 * input_image[p24 * 3] + 4 * input_image[p25 * 3] +
                                       5 * input_image[p31 * 3] + 12 * input_image[p32 * 3]  + 15 * input_image[p33 * 3] + 12 * input_image[p34 * 3] + 5 * input_image[p35 * 3] +
                                       4 * input_image[p41 * 3] +  9 * input_image[p42 * 3]  + 12 * input_image[p43 * 3] +  9 * input_image[p44 * 3] + 4 * input_image[p45 * 3] +
                                       2 * input_image[p51 * 3] +  4 * input_image[p52 * 3]  +  5 * input_image[p53 * 3] +  4 * input_image[p54 * 3] + 2 * input_image[p55 * 3] ) / 159;
        output_image[offset * 3 + 1] = (
                                           2 * input_image[p11 * 3 + 1] +  4 * input_image[p12 * 3 + 1]  +  5 * input_image[p13 * 3 + 1] +  4 * input_image[p14 * 3 + 1] + 2 * input_image[p15 * 3 + 1] +
                                           4 * input_image[p21 * 3 + 1] +  9 * input_image[p22 * 3 + 1]  + 12 * input_image[p23 * 3 + 1] +  9 * input_image[p24 * 3 + 1] + 4 * input_image[p25 * 3 + 1] +
                                           5 * input_image[p31 * 3 + 1] + 12 * input_image[p32 * 3 + 1]  + 15 * input_image[p33 * 3 + 1] + 12 * input_image[p34 * 3 + 1] + 5 * input_image[p35 * 3 + 1] +
                                           4 * input_image[p41 * 3 + 1] +  9 * input_image[p42 * 3 + 1]  + 12 * input_image[p43 * 3 + 1] +  9 * input_image[p44 * 3 + 1] + 4 * input_image[p45 * 3 + 1] +
                                           2 * input_image[p51 * 3 + 1] +  4 * input_image[p52 * 3 + 1]  +  5 * input_image[p53 * 3 + 1] +  4 * input_image[p54 * 3 + 1] + 2 * input_image[p55 * 3 + 1] ) / 159;
        output_image[offset * 3 + 2] = (
                                           2 * input_image[p11 * 3 + 2] +  4 * input_image[p12 * 3 + 2]  +  5 * input_image[p13 * 3 + 2] +  4 * input_image[p14 * 3 + 2] + 2 * input_image[p15 * 3 + 2] +
                                           4 * input_image[p21 * 3 + 2] +  9 * input_image[p22 * 3 + 2]  + 12 * input_image[p23 * 3 + 2] +  9 * input_image[p24 * 3 + 2] + 4 * input_image[p25 * 3 + 2] +
                                           5 * input_image[p31 * 3 + 2] + 12 * input_image[p32 * 3 + 2]  + 15 * input_image[p33 * 3 + 2] + 12 * input_image[p34 * 3 + 2] + 5 * input_image[p35 * 3 + 2] +
                                           4 * input_image[p41 * 3 + 2] +  9 * input_image[p42 * 3 + 2]  + 12 * input_image[p43 * 3 + 2] +  9 * input_image[p44 * 3 + 2] + 4 * input_image[p45 * 3 + 2] +
                                           2 * input_image[p51 * 3 + 2] +  4 * input_image[p52 * 3 + 2]  +  5 * input_image[p53 * 3 + 2] +  4 * input_image[p54 * 3 + 2] + 2 * input_image[p55 * 3 + 2] ) / 159;

        //}
    } else {
        if (offset < width * height) {
            output_image[offset * 3]     = input_image[offset * 3];
            output_image[offset * 3 + 1] = input_image[offset * 3 + 1];
            output_image[offset * 3 + 2] = input_image[offset * 3 + 2];
        }
    }
}
__global__ void kernel_Media(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;

    if (x < width - 3 && x > 3 && y < height - 3 && y > 3) {
        int p11 = offset - 2 * width - 2;
        int p12 = offset - 2 * width - 1;
        int p13 = offset - 2 * width;
        int p14 = offset - 2 * width + 1;
        int p15 = offset - 2 * width + 2;

        int p21 = offset - width - 2;
        int p22 = offset - width - 1;
        int p23 = offset - width;
        int p24 = offset - width + 1;
        int p25 = offset - width + 2;

        int p31 = offset - 2;
        int p32 = offset - 1;
        int p33 = offset;
        int p34 = offset + 1;
        int p35 = offset + 2;

        int p41 = offset + width - 2;
        int p42 = offset + width - 1;
        int p43 = offset + width;
        int p44 = offset + width + 1;
        int p45 = offset + width + 2;

        int p51 = offset + 2 * width - 2;
        int p52 = offset + 2 * width - 1;
        int p53 = offset + 2 * width;
        int p54 = offset + 2 * width + 1;
        int p55 = offset + 2 * width + 2;

        output_image[offset * 3] = (
                                       1 * input_image[p11 * 3] + 1 * input_image[p12 * 3]  + 1 * input_image[p13 * 3] + 1 * input_image[p14 * 3] + 1 * input_image[p15 * 1] +
                                       1 * input_image[p21 * 3] + 1 * input_image[p22 * 3]  + 1 * input_image[p23 * 3] + 1 * input_image[p24 * 3] + 1 * input_image[p25 * 1] +
                                       1 * input_image[p31 * 3] + 1 * input_image[p32 * 3]  + 1 * input_image[p33 * 3] + 1 * input_image[p34 * 3] + 1 * input_image[p35 * 1] +
                                       1 * input_image[p41 * 3] + 1 * input_image[p42 * 3]  + 1 * input_image[p43 * 3] + 1 * input_image[p44 * 3] + 1 * input_image[p45 * 1] +
                                       1 * input_image[p51 * 3] + 1 * input_image[p52 * 3]  + 1 * input_image[p53 * 3] + 1 * input_image[p54 * 3] + 1 * input_image[p55 * 1] ) / 25;
        output_image[offset * 3 + 1] = (
                                           1 * input_image[p11 * 3 + 1] + 1 * input_image[p12 * 3 + 1]  + 1 * input_image[p13 * 3 + 1] + 1 * input_image[p14 * 3 + 1] + 1 * input_image[p15 * 3 + 1] +
                                           1 * input_image[p21 * 3 + 1] + 1 * input_image[p22 * 3 + 1]  + 1 * input_image[p23 * 3 + 1] + 1 * input_image[p24 * 3 + 1] + 1 * input_image[p25 * 3 + 1] +
                                           1 * input_image[p31 * 3 + 1] + 1 * input_image[p32 * 3 + 1]  + 1 * input_image[p33 * 3 + 1] + 1 * input_image[p34 * 3 + 1] + 1 * input_image[p35 * 3 + 1] +
                                           1 * input_image[p41 * 3 + 1] + 1 * input_image[p42 * 3 + 1]  + 1 * input_image[p43 * 3 + 1] + 1 * input_image[p44 * 3 + 1] + 1 * input_image[p45 * 3 + 1] +
                                           1 * input_image[p51 * 3 + 1] + 1 * input_image[p52 * 3 + 1]  + 1 * input_image[p53 * 3 + 1] + 1 * input_image[p54 * 3 + 1] + 1 * input_image[p55 * 3 + 1] ) / 25;
        output_image[offset * 3 + 2] = (
                                           1 * input_image[p11 * 3 + 2] + 1 * input_image[p12 * 3 + 2]  + 1 * input_image[p13 * 3 + 2] + 1 * input_image[p14 * 3 + 2] + 1 * input_image[p15 * 3 + 2] +
                                           1 * input_image[p21 * 3 + 2] + 1 * input_image[p22 * 3 + 2]  + 1 * input_image[p23 * 3 + 2] + 1 * input_image[p24 * 3 + 2] + 1 * input_image[p25 * 3 + 2] +
                                           1 * input_image[p31 * 3 + 2] + 1 * input_image[p32 * 3 + 2]  + 1 * input_image[p33 * 3 + 2] + 1 * input_image[p34 * 3 + 2] + 1 * input_image[p35 * 3 + 2] +
                                           1 * input_image[p41 * 3 + 2] + 1 * input_image[p42 * 3 + 2]  + 1 * input_image[p43 * 3 + 2] + 1 * input_image[p44 * 3 + 2] + 1 * input_image[p45 * 3 + 2] +
                                           1 * input_image[p51 * 3 + 2] + 1 * input_image[p52 * 3 + 2]  + 1 * input_image[p53 * 3 + 2] + 1 * input_image[p54 * 3 + 2] + 1 * input_image[p55 * 3 + 2] ) / 25;

    } else {
        if (offset < width * height) {
            output_image[offset * 3]     = input_image[offset * 3];
            output_image[offset * 3 + 1] = input_image[offset * 3 + 1];
            output_image[offset * 3 + 2] = input_image[offset * 3 + 2];
        }
    }
}
__device__ void sort(int* arr, int n) {
    int temp;
    int i, j;
    for (i = 0; i < n - 1; i++) {
        for (j = 0; j < n - i - 1; j++) {
            if (arr[j] > arr[j + 1]) {
                temp = arr[j];
                arr[j] = arr[j + 1];
                arr[j + 1] = temp;
            }
        }
    }
}
__global__ void kernel_Sobel(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    if (x < width - 1 && x > 2 && y < height - 1 && y > 2) {
        int p11 = offset - width - 1;
        int p12 = offset - width;
        int p13 = offset - width + 1;

        int p21 = offset - 1;
        int p22 = offset;
        int p23 = offset + 1;

        int p31 = offset + width - 1;
        int p32 = offset + width;
        int p33 = offset + width + 1;

        int auxInte1 = (    input_image[p11 * 3]  -   input_image[p13 * 3] +
                            2 * input_image[p21 * 3]  - 2 * input_image[p23 * 3] +
                            input_image[p31 * 3]  -   input_image[p33 * 3]) / 4;

        int auxInte2 = (   -input_image[p11 * 3] - 2 * input_image[p12 * 3] - input_image[p13 * 3] +
                           input_image[p31 * 3] + 2 * input_image[p32 * 3] + input_image[p33 * 3]) / 4;

        auxInte1 =  sqrtf((auxInte1 * auxInte1) + (auxInte2 * auxInte2))  ;

        if (auxInte1 > 255 ) {
            output_image[offset * 3]   = 255;
            output_image[offset * 3 + 1] = 255;
            output_image[offset * 3 + 2] = 255;
        } else if (auxInte1 < 0 ) {
            output_image[offset * 3]   = 0;
            output_image[offset * 3 + 1] = 0;
            output_image[offset * 3 + 2] = 0;
        }
        else {
            output_image[offset * 3]   = auxInte1;
            output_image[offset * 3 + 1] = auxInte1;
            output_image[offset * 3 + 2] = auxInte1;
        }
    } else {
        if (offset < width * height) {
            output_image[offset * 3]     = input_image[offset * 3];
            output_image[offset * 3 + 1] = input_image[offset * 3 + 1];
            output_image[offset * 3 + 2] = input_image[offset * 3 + 2];
        }
    }
}
__global__ void kernel_Mediana(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    if (x < width - 1 && x > 2 && y < height - 1 && y > 2) {


        int p11 = offset - width - 1;
        int p12 = offset - width;
        int p13 = offset - width + 1;

        int p21 = offset - 1;
        int p22 = offset;
        int p23 = offset + 1;

        int p31 = offset + width - 1;
        int p32 = offset + width;
        int p33 = offset + width + 1;

        int r[9];
        int g[9];
        int b[9];
        r[0] = input_image[p11 * 3];
        r[1] = input_image[p12 * 3];
        r[2] = input_image[p13 * 3];
        r[3] = input_image[p21 * 3];
        r[4] = input_image[p22 * 3];
        r[5] = input_image[p23 * 3];
        r[6] = input_image[p31 * 3];
        r[7] = input_image[p32 * 3];
        r[8] = input_image[p33 * 3];

        g[0] = input_image[p11 * 3 + 1];
        g[1] = input_image[p12 * 3 + 1];
        g[2] = input_image[p13 * 3 + 1];
        g[3] = input_image[p21 * 3 + 1];
        g[4] = input_image[p22 * 3 + 1];
        g[5] = input_image[p23 * 3 + 1];
        g[6] = input_image[p31 * 3 + 1];
        g[7] = input_image[p32 * 3 + 1];
        g[8] = input_image[p33 * 3 + 1];

        b[0] = input_image[p11 * 3 + 2];
        b[1] = input_image[p12 * 3 + 2];
        b[2] = input_image[p13 * 3 + 2];
        b[3] = input_image[p21 * 3 + 2];
        b[4] = input_image[p22 * 3 + 2];
        b[5] = input_image[p23 * 3 + 2];
        b[6] = input_image[p31 * 3 + 2];
        b[7] = input_image[p32 * 3 + 2];
        b[8] = input_image[p33 * 3 + 2];
        sort(r, 9);
        sort(g, 9);
        sort(b, 9);
        //if ( offset < height * width) {
        output_image[offset * 3] = r[5];
        output_image[offset * 3 + 1] = g[5];
        output_image[offset * 3 + 2] = b[5];
        //}
    } else {
        if (offset < width * height) {
            output_image[offset * 3]     = input_image[offset * 3];
            output_image[offset * 3 + 1] = input_image[offset * 3 + 1];
            output_image[offset * 3 + 2] = input_image[offset * 3 + 2];
        }
    }
}
__global__ void kernel_Equalizar(unsigned char *input_image, unsigned char *output_image, int width, int height, int r, int g, int b) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int new_r;
    int new_g;
    int new_b;
    if (offset < width * height) {
        new_r = input_image[offset * 3] + r;
        new_g = input_image[offset * 3 + 1] + g;
        new_b = input_image[offset * 3 + 2] + b;
        if (new_r < 0 ) {
            output_image[offset * 3] = 0;
        } else if (new_r > 255 ) {
            output_image[offset * 3] = 255;
        } else {
            output_image[offset * 3] = new_r;
        }
        if (new_g < 0 ) {
            output_image[offset * 3 + 1] = 0;
        } else if (new_g > 255 ) {
            output_image[offset * 3 + 1] = 255;
        } else {
            output_image[offset * 3 + 1] = new_g;
        }
        if (new_b < 0 ) {
            output_image[offset * 3 + 2] = 0;
        } else if (new_b > 255 ) {
            output_image[offset * 3 + 2] = 255;
        } else {
            output_image[offset * 3 + 2] = new_b;
        }
    }
}
__global__ void kernel_Equalizar_2(unsigned char *input_image, unsigned char *output_image, int width, int height, int r, float r_factor, int g, float g_factor, int b, float b_factor) {
    int temp = width;
    width = height;
    height = temp;
    int new_r;
    int new_g;
    int new_b;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < width * height) {
        new_r = (input_image[offset * 3] - r) * r_factor;
        new_g = (input_image[offset * 3 + 1] - g) * g_factor;
        new_b = (input_image[offset * 3 + 2] - b) * b_factor;
        if (new_r < 0 ) {
            output_image[offset * 3] = 0;
        } else if (new_r > 255 ) {
            output_image[offset * 3] = 255;
        } else {
            output_image[offset * 3] = new_r;
        }
        if (new_g < 0 ) {
            output_image[offset * 3 + 1] = 0;
        } else if (new_g > 255 ) {
            output_image[offset * 3 + 1] = 255;
        } else {
            output_image[offset * 3 + 1] = new_g;
        }
        if (new_b < 0 ) {
            output_image[offset * 3 + 2] = 0;
        } else if (new_b > 255 ) {
            output_image[offset * 3 + 2] = 255;
        } else {
            output_image[offset * 3 + 2] = new_b;
        }
    }
}
__global__ void kernel_rotate(unsigned char * input_image, unsigned char * output_image, int width, int height, float sn, float cs) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    int xc = width / 2;
    int yc = height / 2;
    int newx = ((float)x - xc) * cs - ((float)y - yc) * sn + xc;
    int newy = ((float)x - xc) * sn + ((float)y - yc) * cs + yc;
    int new_offset = newx  + newy * width;
    if(x < width && y<height && newx < width && newx > 0 && newy < height && newy > 0){
        output_image[offset * 3 + 0] = input_image[new_offset * 3 + 0];
        output_image[offset * 3 + 1] = input_image[new_offset * 3 + 1];
        output_image[offset * 3 + 2] = input_image[new_offset * 3 + 2];
    }
}
__global__ void kernel_resize(unsigned char *input_image, unsigned char *output_image, int width, int height, int width_out, int height_out) {
    int temp = width;
    width = height;
    height = temp;
    temp = width_out;
    width_out = height_out;
    height_out = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width_out;
    int y = (offset - x) / width_out;
    float scale = width_out * 1.0 / width;
    int newx = x / scale;
    int newy = y / scale;
    int new_offset;

    if ( offset < height_out * width_out) {
        new_offset = newx  + newy * width;
        output_image[offset * 3 + 0] = input_image[new_offset * 3 + 0];
        output_image[offset * 3 + 1] = input_image[new_offset * 3 + 1];
        output_image[offset * 3 + 2] = input_image[new_offset * 3 + 2];
    }
}
__global__ void kernel_pattern_search(unsigned char *input_image, unsigned char *output_image, unsigned char * pattern_image, int width, int height, int width_pattern, int height_pattern,double percent) {
    int temp = width;
    width = height;
    height = temp;
    temp = width_pattern;
    width_pattern = height_pattern;
    height_pattern = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int off_input = offset;
    int off_pattern = 0;
    int hit_count = 0;
    if (offset < width * height - width_pattern * height_pattern) {
        for (int j = 0; j < height_pattern; j++) {
            for ( int i = 0; i < width_pattern; i++) {
                if ( input_image[off_input * 3]     == pattern_image[off_pattern * 3 ] &&
                        input_image[off_input * 3 + 1] == pattern_image[off_pattern * 3 + 1] &&
                        input_image[off_input * 3 + 2] == pattern_image[off_pattern * 3 + 2] ) {
                    hit_count++;
                }
                off_input++;
                off_pattern++;
            }
            off_input += width - width_pattern;
        }
        off_input = offset;
        if (hit_count >= width_pattern * height_pattern * percent) {
            for (int j = 0; j < height_pattern; j++) {
                for ( int i = 0; i < width_pattern; i++) {
                    output_image[off_input * 3]     =   0 + 0.5*input_image[off_input * 3]; //pattern_image[off_pattern * 3 ] ;
                    output_image[off_input * 3 + 1] = 128 + 0.5*input_image[off_input * 3 + 1]; //pattern_image[off_pattern * 3 + 1] ;
                    output_image[off_input * 3 + 2] =   0 + 0.5*input_image[off_input * 3 + 2]; //pattern_image[off_pattern * 3 + 2] ;
                    off_input++;
                }
                off_input += width - width_pattern;
            }
        }
    }
}
__global__ void kernel_image_to_gray(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int new_c;
    if (offset < width * height) {
        new_c = (   input_image[offset * 3] +
                    input_image[offset * 3 + 1] +
                    input_image[offset * 3 + 2] ) / 3;
        output_image[offset * 3]     = new_c;
        output_image[offset * 3 + 1] = new_c;
        output_image[offset * 3 + 2] = new_c;
    }
}
__global__ void kernel_image_erosion(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    int new_c = 255;
    if (x < width - 1 && x > 2 && y < height - 1 && y > 2) {
        int p12 = offset - width;
        int p21 = offset - 1;
        int p22 = offset;
        int p23 = offset + 1;
        int p32 = offset + width;

        int c12 = ( input_image[p12 * 3] +
                    input_image[p12 * 3 + 1] +
                    input_image[p12 * 3 + 2] ) / 3 / 128;
        int c21 = ( input_image[p21 * 3] +
                    input_image[p21 * 3 + 1] +
                    input_image[p21 * 3 + 2] ) / 3 / 128;
        int c22 = ( input_image[p22 * 3] +
                    input_image[p22 * 3 + 1] +
                    input_image[p22 * 3 + 2] ) / 3 / 128;
        int c23 = ( input_image[p23 * 3] +
                    input_image[p23 * 3 + 1] +
                    input_image[p23 * 3 + 2] ) / 3 / 128;
        int c32 = ( input_image[p32 * 3] +
                    input_image[p32 * 3 + 1] +
                    input_image[p32 * 3 + 2] ) / 3 / 128;
        int cant_hits = 5 - (c12 + c21 + c22 + c23 + c32);

        if (cant_hits == 5) {
            new_c = 0;
        }
        int umbral = 128;
        if ( offset < height * width) {
            output_image[offset * 3]     = new_c;
            output_image[offset * 3 + 1] = new_c;
            output_image[offset * 3 + 2] = new_c;
        }
    } else {
        if (offset < width * height) {
            new_c = (   input_image[offset * 3] +
                        input_image[offset * 3 + 1] +
                        input_image[offset * 3 + 2] ) / 3 / 128 * 255;
            output_image[offset * 3]     = new_c;
            output_image[offset * 3 + 1] = new_c;
            output_image[offset * 3 + 2] = new_c;
        }
    }
}
__global__ void kernel_image_dilatation(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    int new_c = 255;

    if (x < width - 1 && x > 2 && y < height - 1 && y > 2) {
        int p12 = offset - width;
        int p21 = offset - 1;
        int p22 = offset;
        int p23 = offset + 1;
        int p32 = offset + width;

        int c12 = ( input_image[p12 * 3] +
                    input_image[p12 * 3 + 1] +
                    input_image[p12 * 3 + 2] ) / 3 / 128;
        int c21 = ( input_image[p21 * 3] +
                    input_image[p21 * 3 + 1] +
                    input_image[p21 * 3 + 2] ) / 3 / 128;
        int c22 = ( input_image[p22 * 3] +
                    input_image[p22 * 3 + 1] +
                    input_image[p22 * 3 + 2] ) / 3 / 128;
        int c23 = ( input_image[p23 * 3] +
                    input_image[p23 * 3 + 1] +
                    input_image[p23 * 3 + 2] ) / 3 / 128;
        int c32 = ( input_image[p32 * 3] +
                    input_image[p32 * 3 + 1] +
                    input_image[p32 * 3 + 2] ) / 3 / 128;
        int cant_hits = 5 - ( c12 + c21 + c22 + c23 + c32);
        if (cant_hits > 1) {
            new_c = 0;
        }
        int umbral = 128;
        if ( offset < height * width) {
            output_image[offset * 3]     = new_c;
            output_image[offset * 3 + 1] = new_c;
            output_image[offset * 3 + 2] = new_c;
        }
    } else {
        if (offset < width * height) {
            new_c = (   input_image[offset * 3] +
                        input_image[offset * 3 + 1] +
                        input_image[offset * 3 + 2] ) / 3 / 128 * 255;
            output_image[offset * 3]     = new_c;
            output_image[offset * 3 + 1] = new_c;
            output_image[offset * 3 + 2] = new_c;
        }
    }
}
__global__ void kernel_image_to_bn(unsigned char *input_image, unsigned char *output_image, int width, int height) {
    int temp = width;
    width = height;
    height = temp;
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int new_c;
    if (offset < width * height) {
        new_c = (   input_image[offset * 3] +
                    input_image[offset * 3 + 1] +
                    input_image[offset * 3 + 2] ) / 3 / 128 * 255;
        /*if (new_c > 128) {
            new_c = 255;
        } else {
            new_c = 0;
        }*/
        output_image[offset * 3]     = new_c;
        output_image[offset * 3 + 1] = new_c;
        output_image[offset * 3 + 2] = new_c;
    }
}
void loadArrayFromMat(Mat& imagen, unsigned char* array, int width, int height) {
    int i = 0;
    for (int w = 0; w < width; w++) {
        for (int h = 0; h < height; h++) {
            MatPixel& pixel = imagen.at<MatPixel>(h, w);
            array[i++] = pixel.r;
            array[i++] = pixel.g;
            array[i++] = pixel.b;
        }
    }
}
void loadMatFromArray(Mat& imagen, unsigned char* array, int width, int height) {
    int i = 0;
    for (int w = 0; w < width; w++) {
        for (int h = 0; h < height; h++) {
            MatPixel& pixel = imagen.at<MatPixel>(h, w);
            pixel.r = array[i++];
            pixel.g = array[i++];
            pixel.b = array[i++];
        }
    }
}
vector<complex<double>> fft(vector<complex<double>>& a) {
    int n = a.size();

    if (n == 1)
        return vector<complex<double>>(1, a[0]);

    vector<complex<double>> w(n);
    for (int i = 0; i < n; i++) {
        double alpha = 2 * M_PI * i / n;
        w[i] = complex<double>(cos(alpha), sin(alpha));
    }

    vector<complex<double>> even_elements(n / 2), odd_elements(n / 2);
    for (int i = 0; i < n / 2; i++) {
        even_elements[i] = a[i * 2];
        odd_elements[i] = a[i * 2 + 1];
    }

    vector<complex<double>> y0 = fft(even_elements);
    vector<complex<double>> y1 = fft(odd_elements);

    vector<complex<double>> y(n);

    for (int k = 0; k < n / 2; k++) {
        y[k] = y0[k] + w[k] * y1[k];
        y[k + n / 2] = y0[k] - w[k] * y1[k];
    }
    return y;
}
void EqualizarHistograma(unsigned char *cpu_input_image, unsigned char *input_image, unsigned char* output_image, int width, int height) {
    //int temp = width;
    //width = height;
    //height = temp;
    int *r = new int[256];
    int *g = new int[256];
    int *b = new int[256];
    for (int i = 0; i < 256; i++) {
        r[i] = 0;
        g[i] = 0;
        b[i] = 0;
    }
    int new_r, new_g, new_b;
    for (int i = 0; i < width * height; i++) {
        new_r = cpu_input_image[i * 3];
        new_g = cpu_input_image[i * 3 + 1] ;
        new_b = cpu_input_image[i * 3 + 2] ;
        if (new_r < 0 ) {
            r[0]++;
        } else if (new_r > 255 ) {
            r[255]++;
        } else {
            r[new_r]++;
        }
        if (new_g < 0 ) {
            b[0]++;
        } else if (new_g > 255 ) {
            b[255]++;
        } else {
            b[new_g]++;
        }
        if (new_b < 0 ) {
            b[0]++;
        } else if (new_b > 255 ) {
            b[255]++;
        } else {
            b[new_b]++;
        }
    }
    int r_max = 0, r_min = 255;
    int g_max = 0, g_min = 255;
    int b_max = 0, b_min = 255;
    for (int i = 0; i < 10; i++) {
        if (r[i] > 0) {
            if (i < r_min)r_min = i;
            if (i > r_max)r_max = i;
        }
        if (g[i] > 0) {
            if (i < g_min)g_min = i;
            if (i > g_max)g_max = i;
        }
        if (b[i] > 0) {
            if (i < b_min)b_min = i;
            if (i > b_max)b_max = i;
        }
    }
    float r_factor = 256;
    if (r_max - r_min > 0)
        r_factor /= (r_max - r_min);
    else r_factor = 1;
    float g_factor = 256;
    if (g_max - g_min > 0)
        g_factor /= (g_max - g_min);
    else g_factor = 1;
    float b_factor = 256;
    if (b_max - b_min > 0)
        b_factor /= (b_max - b_min);
    else b_factor = 1;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    //hipEventRecord(start);
    cout << "running kernel" << endl;
    kernel_Equalizar_2 <<< gridDims, blockDims>>>(input_image, output_image, width, height, r_min, r_factor, g_min, g_factor, b_min, b_factor);
    cout << r_min << " " << g_min << " " << b_min << endl;
    //kernel_Equalizar<<< gridDims, blockDims>>>(input_image, output_image, width, height,1,1,1);
    cout << "Finish kernel " << endl;
    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //float milliseconds = 0;
    //hipEventElapsedTime(&milliseconds, start, stop);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
}
void applyKernelGaussiano(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_Gaussiano <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
    //kernel_Gaussiano<<<gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height);
    //kernel_Equalizar <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, r, g, b);
    //kernel_rotate <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, sn, cs);
}
void applyKernelMedia(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_Media <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
}
void applyKernelMediana(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_Mediana <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
}
void applyKernelGray(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_image_to_gray <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
}
void applyKernelBinary(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    //hipMemcpy( gpu_image_output, cpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    //hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_image_to_bn <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
}
void applyKernelEqualizar(unsigned char* input_image, unsigned char* output_image, int width, int height, int r, int g, int b) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_Equalizar <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, r, g, b);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp   = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input  = gpu_image_temp;
}
void applyKernelErosion(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_image_erosion <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp   = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input  = gpu_image_temp;
}
void applyKernelSobel(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_Sobel <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp   = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input  = gpu_image_temp;
}
void action_rotate(unsigned char* input_image, unsigned char* output_image, int width, int height,float sn,float cs){
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_rotate <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, sn, cs);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    /*gpu_image_temp   = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input  = gpu_image_temp;*/
}
void applyKernelDilatation(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    //hipMemcpy( gpu_image_output, cpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    //hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_image_dilatation <<< gridDims, blockDims>>>(input_image, output_image, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    gpu_image_temp = gpu_image_output;
    gpu_image_output = gpu_image_input;
    gpu_image_input = gpu_image_temp;
}
void action_search_pattern(/*unsigned char* input_image, unsigned char* output_image,unsigned char* patter_image,*/ int width, int height, int width_pattern, int height_pattern,double percent) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel_pattern_search <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, gpu_image_aux, width, height, width_pattern, height_pattern,percent/100.0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << endl;
    //gpu_image_temp = gpu_image_output;
    //gpu_image_output = gpu_image_input;
    //gpu_image_input = gpu_image_temp;
    hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
    loadMatFromArray(image_output, cpu_image_output, width, height);
    //kernel_Gaussiano<<<gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height);
    //kernel_Equalizar <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, r, g, b);
    //kernel_rotate <<< gridDims, blockDims>>>(gpu_image_input, gpu_image_output, width, height, sn, cs);
}
void generateNoise(unsigned char* input_image, int width, int height, float percent) {
    int noise_pixels = width * height * percent / 100;
    int n_w;
    int n_h;
    int offset;
    for (int i = 0; i < noise_pixels; i++) {
        n_w = rand() % width;
        n_h = rand() % height;
        offset = n_w * width + n_h;
        if (offset < width * height) {
            input_image[offset * 3] = 0;
            input_image[offset * 3 + 1] = 0;
            input_image[offset * 3 + 2] = 0;
        }
    }
    hipMemcpy( cpu_image_input, gpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
}
Mat calcHist(Mat& image) {
    int histSize = 256;

    /// Set the ranges ( for B,G,R) )
    float range[] = { 0, 256 } ;
    const float* histRange = { range };
    bool uniform = true; bool accumulate = false;
    vector<Mat> bgr_planes;
    split( image, bgr_planes );
    Mat b_hist, g_hist, r_hist;
    calcHist( &bgr_planes[0], 1, 0, Mat(), b_hist, 1, &histSize, &histRange, uniform, accumulate );
    calcHist( &bgr_planes[1], 1, 0, Mat(), g_hist, 1, &histSize, &histRange, uniform, accumulate );
    calcHist( &bgr_planes[2], 1, 0, Mat(), r_hist, 1, &histSize, &histRange, uniform, accumulate );
    int hist_w = 180; int hist_h = 150;
    int bin_w = cvRound( (double) hist_w / histSize );

    Mat histImage( hist_h, hist_w, CV_8UC3, Scalar( 0, 0, 0) );

    /// Normalize the result to [ 0, histImage.rows ]
    normalize(b_hist, b_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat() );
    normalize(g_hist, g_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat() );
    normalize(r_hist, r_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat() );

    /// Draw for each channel
    for ( int i = 1; i < histSize; i++ ) {
        line( histImage, Point( bin_w * (i - 1), hist_h - cvRound(b_hist.at<float>(i - 1)) ) ,
              Point( bin_w * (i), hist_h - cvRound(b_hist.at<float>(i)) ),
              Scalar( 255, 0, 0), 2, 8, 0  );
        line( histImage, Point( bin_w * (i - 1), hist_h - cvRound(g_hist.at<float>(i - 1)) ) ,
              Point( bin_w * (i), hist_h - cvRound(g_hist.at<float>(i)) ),
              Scalar( 0, 255, 0), 2, 8, 0  );
        line( histImage, Point( bin_w * (i - 1), hist_h - cvRound(r_hist.at<float>(i - 1)) ) ,
              Point( bin_w * (i), hist_h - cvRound(r_hist.at<float>(i)) ),
              Scalar( 0, 0, 255), 2, 8, 0  );
    }
    return histImage;
}
Mat calcFastFourierTransform(Mat& image, int width, int height) {
    Mat gray_image;
    cvtColor( image, gray_image, CV_BGR2GRAY );
    //int i = 0;
    vector<complex<double>> cols(width);
    vector<complex<double>> rows(height);
    vector<complex<double>> temp;
    vector<vector<complex<double>>> fftMat;
    for (int h = 0; h < height; h++) {
        for (int w = 0; w < width; w++) {
            cols[w] = {(double)gray_image.at<uchar>(h, w), 0.0};
        }
        temp = fft(cols);
        fftMat.push_back(temp);
    }

    for (int w = 0; w < width; w++) {
        for (int h = 0; h < height; h++) {
            cols[h] = fftMat[h][w];
        }
        temp = fft(cols);
        for (int h = 0; h < height; h++) {
            gray_image.at<uchar>(h, w) = log(abs(temp[h]) + 1);
        }
    }

    gray_image = gray_image(Rect(0, 0, gray_image.cols & -2, gray_image.rows & -2));

    int cx = gray_image.cols / 2;
    int cy = gray_image.rows / 2;

    Mat q0(gray_image, Rect(0, 0, cx, cy));
    Mat q1(gray_image, Rect(cx, 0, cx, cy));
    Mat q2(gray_image, Rect(0, cy, cx, cy));
    Mat q3(gray_image, Rect(cx, cy, cx, cy));

    Mat tmp;
    q0.copyTo(tmp);
    q3.copyTo(q0);
    tmp.copyTo(q3);

    q1.copyTo(tmp);
    q2.copyTo(q1);
    tmp.copyTo(q2);

    normalize(gray_image, gray_image, 0, 255, CV_MINMAX);

    return gray_image;
}
void actionShowFFT(Mat& image, int width, int height) {
    Mat image_FFT =  calcFastFourierTransform(image, width, height);
    imshow("Fast Fourier Transform", image_FFT);
}
void action_open_file(Mat &image) {
    FILE *in;
    if (!(in = popen("zenity  --title=\"Seleccionar imagen\" --file-selection", "r"))) {
        image_loaded = false;
        return;
    }
    image_loaded = true;
    char buff[512];
    file_path = "";
    while (fgets(buff, sizeof(buff), in) != NULL) {
        file_path += buff;
    }
    pclose(in);

    //remove the "\n"
    file_path.erase(std::remove(file_path.begin(), file_path.end(), '\n'), file_path.end());

    // path + filename + format
    image = imread(file_path);
    int width = image.cols;
    int height = image.rows;

    delete cpu_image_input;
    delete cpu_image_output;
    cpu_image_output = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    cpu_image_input = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    hipFree(gpu_image_output);
    hipFree(gpu_image_input);
    hipMalloc( (void**)&gpu_image_output, width * height * 3 * sizeof(unsigned char));
    hipMalloc( (void**)&gpu_image_input, width * height * 3 * sizeof(unsigned char) );

    loadArrayFromMat(image, cpu_image_input, width, height);
    ////loadMatFromArray()
    //
    //image_output = Mat(height, width, CV_8UC3, Scalar(255, 0, 0));
    //image_output = image_input;
    //hipMemcpy( gpu_image_output, cpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_image_output, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    blockDims = dim3(512, 1, 1);
    gridDims = dim3((unsigned int) ceil((double)(width * height * 3 / blockDims.x)), 1, 1 );
    //int rotate_angle = 0;
}
void action_open_file_aux(Mat &image) {
    FILE *in;
    if (!(in = popen("zenity  --title=\"Seleccionar imagen\" --file-selection", "r"))) {
        return;
    }

    char buff[512];
    string selectFile = "";
    while (fgets(buff, sizeof(buff), in) != NULL) {
        selectFile += buff;
    }
    pclose(in);

    //remove the "\n"
    selectFile.erase(std::remove(selectFile.begin(), selectFile.end(), '\n'), selectFile.end());

    // path + filename + format
    image = imread(selectFile);
    int width = image.cols;
    int height = image.rows;

    delete cpu_image_aux;
    //delete cpu_image_output;
    cpu_image_aux = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    //cpu_image_input = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    hipFree(gpu_image_aux);
    //hipFree(gpu_image_input);
    hipMalloc( (void**)&gpu_image_aux, width * height * 3 * sizeof(unsigned char));
    //hipMalloc( (void**)&gpu_image_input, width * height * 3 * sizeof(unsigned char) );

    loadArrayFromMat(image, cpu_image_aux, width, height);
    ////loadMatFromArray()
    //
    //image_output = Mat(height, width, CV_8UC3, Scalar(255, 0, 0));
    //image_output = image_input;
    hipMemcpy( gpu_image_aux, cpu_image_aux, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    //hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    //blockDims = dim3(512, 1, 1);
    //gridDims = dim3((unsigned int) ceil((double)(width * height * 3 / blockDims.x)), 1, 1 );
    //int rotate_angle = 0;
}
void reload_image(Mat &image) {
    // path + filename + format
    image = imread(file_path);
    int width = image.cols;
    int height = image.rows;

    delete cpu_image_input;
    delete cpu_image_output;
    cpu_image_output = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    cpu_image_input = (unsigned char*)malloc(sizeof(unsigned char) * height * width * 3);
    hipFree(gpu_image_output);
    hipFree(gpu_image_input);
    hipMalloc( (void**)&gpu_image_output, width * height * 3 * sizeof(unsigned char));
    hipMalloc( (void**)&gpu_image_input, width * height * 3 * sizeof(unsigned char) );

    loadArrayFromMat(image, cpu_image_input, width, height);
    ////loadMatFromArray()
    //
    //image_output = Mat(height, width, CV_8UC3, Scalar(255, 0, 0));
    //image_output = image_input;
    //hipMemcpy( gpu_image_output, cpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_image_output, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
    blockDims = dim3(512, 1, 1);
    gridDims = dim3((unsigned int) ceil((double)(width * height * 3 / blockDims.x)), 1, 1 );
    //int rotate_angle = 0;
}
int main( int argc, char** argv ) {
    char * message = "";
    unsigned int theColor = 0xff0000;
    Mat frame = cv::Mat(980, 1170, CV_8UC3);
    Mat image_histogram, image_FFT, image_scale;
    //Mat image_temp = cv::Mat(CANVAS_WIDTH, CANVAS_HEIGHT, CV_8UC3, Scalar(40, 40, 40));

    cv::namedWindow(WINDOW_NAME);
    cvui::init(WINDOW_NAME);
    char tecla;
    double r = 0, g = 0, b = 0;
    double r_l = 0, g_l = 0, b_l = 0;
    double percent = 80;
    int height  = CANVAS_HEIGHT;
    int width   = CANVAS_WIDTH;
    //int rotate ;
    int rotate_angle = 0;
    int rotate_angle_l = 0;
    double scale = 1;
    double scale_l = 1;
    float sn,cs,deg;

    //int degree = 0;
    //float zoom = 1;
    image_canvas = cv::Mat(CANVAS_WIDTH, CANVAS_HEIGHT, CV_8UC3, Scalar(40, 40, 40));

    image_output = cv::Mat(CANVAS_WIDTH, CANVAS_HEIGHT, CV_8UC3);
    image_output = cv::Scalar(40, 40, 40);

    while (1) {
        ROW_ACTUAL = 0;
        frame = cv::Scalar(49, 52, 49);
        if (cpu_image_input) {
            //applyKernelEqualizar(gpu_image_input, gpu_image_output, width, height, r, g, b);
        }
        cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 2 - 5, "CARGAR IMAGEN");
        if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Abrir imagen")) {
            action_open_file(image_output);
            width = image_output.cols;
            height = image_output.rows;
            r = 0, g = 0, b = 0;
            r_l = 0, g_l = 0, b_l = 0;
            rotate_angle = 0,rotate_angle_l=0;
            scale = 1, scale_l = 1;
        }
        if (image_loaded) {
            if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Volver a cargar imagen")) {
                reload_image(image_output);
                width = image_output.cols;
                height = image_output.rows;
                r = 0, g = 0, b = 0;
                r_l = 0, g_l = 0, b_l = 0;
                rotate_angle = 0,rotate_angle_l=0;
                scale = 1, scale_l = 1;
            }
            if (MENU == MENU_TRANSFORM) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 2 - 5, "TRANSFORMACIONES");
                cvui::counter(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, &rotate_angle,5);
                cvui::counter(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, &scale,0.5);
                
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Convertir a grises")) {
                    applyKernelGray(gpu_image_input, gpu_image_output, width, height);
                    r = 0, g = 0, b = 0;
                    r_l = 0, g_l = 0, b_l = 0;
                    //rotate_angle = 0,rotate_angle_l=0;
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Convertir a b/n")) {
                    applyKernelBinary(gpu_image_input, gpu_image_output, width, height);
                    //r = 0, g = 0, b = 0;
                    r = 0, g = 0, b = 0;
                    r_l = 0, g_l = 0, b_l = 0;
                    //rotate_angle = 0,rotate_angle_l=0;
                }
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "TRANSFORMACIONES");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_TRANSFORM;
                }
            }
            if (MENU == MENU_EQUALIZAR) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 5 - 5, "EQUALIZAR");
                cvui::trackbar(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, 150, &r, -250.0, 250.0);
                cvui::trackbar(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, 150, &g, -250.0, 250.0);
                cvui::trackbar(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, 150, &b, -250.0, 250.0);
                /*if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Equalizar")) {
                    EqualizarHistograma(cpu_image_input, gpu_image_input, gpu_image_output, width, height);
                }*/
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "EQUALIZAR");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_EQUALIZAR;
                }
            }
            if (MENU == MENU_CONVOLUCION) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 5 - 5, "CONVOLUCION");
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Sobel")) {
                    applyKernelSobel(gpu_image_input, gpu_image_output, width, height);
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Suavizado Gaussiano")) {
                    applyKernelGaussiano(gpu_image_input, gpu_image_output, width, height);
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Suavizado Media")) {
                    applyKernelMedia(gpu_image_input, gpu_image_output, width, height);
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Suavizado Mediana")) {
                    applyKernelMediana(gpu_image_input, gpu_image_output, width, height);
                    //imshow("result",image_output);
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Agregar ruido")) {
                    generateNoise(cpu_image_input, width, height, 10);
                    loadMatFromArray(image_output, cpu_image_input, width, height);
                    hipMemcpy( gpu_image_input, cpu_image_input, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
                    //imageOutput = Mat(height, width, CV_8UC3, Scalar(255, 0, 0));
                    //imageOutput = imageInput;
                    //hipMemcpy( gpu_image_output, cpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice );
                    //hipMemcpy( cpu_image_output, gpu_image_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
                }
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "CONVOLUCION");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_CONVOLUCION;
                }
            }           
            if (MENU == MENU_FOURIER) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 5 - 5, "FOURIER");
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Fast Fourier Transform")) {
                    actionShowFFT(image_output, width, height);
                }
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "FOURIER");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_FOURIER;
                }
            }
            if (MENU == MENU_MORPH) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 5 - 5, "MORFOLOGIA");
                
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Erosion")) {
                    applyKernelErosion(gpu_image_input, gpu_image_output, width, height);
                    r = 0, g = 0, b = 0;
                    r_l = 0, g_l = 0, b_l = 0;
                    //rotate_angle = 0,rotate_angle_l=0;
                }
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Dilatacion")) {
                    applyKernelDilatation(gpu_image_input, gpu_image_output, width, height);
                    r = 0, g = 0, b = 0;
                    r_l = 0, g_l = 0, b_l = 0;
                    //rotate_angle = 0,rotate_angle_l=0;
                }
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "MORFOLOGIA");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_MORPH;
                }
            }
            if (MENU == MENU_PATTERN_SEARCH) {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL++, COLUMN_2 - 10, ROW_HEIGHT * 5 - 5, "BUSCAR PATRON");
                if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Cargar patron")) {
                    action_open_file_aux(image_aux);
                    pattern_loaded = true;
                }
                if (pattern_loaded) {
                    int aux_width  = image_aux.cols;
                    int aux_height = image_aux.rows;
                    //imshow("Pattern",image_aux);
                    float factor;
                    int desp_x, desp_y;
                    int PATTER_WIDTH  = 180;
                    int PATTER_HEIGHT = 180;
                    if (aux_width < aux_height) {
                        factor = PATTER_HEIGHT * 1.0 / aux_height;
                        desp_x = (PATTER_WIDTH - aux_width * factor ) / 2;
                        desp_y = 0;
                    } else {
                        factor = PATTER_WIDTH  * 1.0 / aux_width;
                        desp_x = 0;
                        desp_y = (PATTER_HEIGHT - aux_height * factor) / 2;
                    }
                    cv::resize(image_aux, image_temp, Size(aux_width * factor, aux_height * factor)); //, interpolation = cv.INTER_CUBIC);
                    //image_temp.copyTo(image_canvas(cv::Rect(desp_x, desp_y, image_temp.cols, image_temp.rows)));
                    cvui::image(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, image_temp);
                    ROW_ACTUAL += 4;
                    cvui::trackbar(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, 150, &percent, 0.0, 100.0);
                    if (cvui::button(frame, COLUMN_1, ROW_HEIGHT * ROW_ACTUAL++, "Buscar en la imagen")) {
                        action_search_pattern(width, height, aux_width, aux_height,percent);
                    }
                }
            } else {
                cvui::window(frame, COLUMN_1 / 2, 5 + ROW_HEIGHT * ROW_ACTUAL, COLUMN_2 - 10, ROW_HEIGHT - 5, "BUSCAR PATRON");
                if (cvui::button(frame, COLUMN_2 - 25, 5 + ROW_HEIGHT * ROW_ACTUAL++, 20, 20, "+")) {
                    MENU = MENU_PATTERN_SEARCH;
                }
            }

            if(rotate_angle !=0){
                hipMemset( gpu_image_output, 0, width * height * 3 * sizeof(unsigned char) );
                deg = rotate_angle * PI / 180;
                sn = sin(deg);
                cs = cos(deg);
                action_rotate(gpu_image_input, gpu_image_output, width, height,sn,cs);
            }
            /*if (r != 0 || g != 0 || b != 0) {
                applyKernelEqualizar(gpu_image_input, gpu_image_output, width, height, r_l - r, g_l - g, b_l - b);
            }*/
            /*if(rotate_angle - rotate_angle_l !=0){
                hipMemset( gpu_image_output, 0, width * height * 3 * sizeof(unsigned char) );
                deg = (rotate_angle -rotate_angle_l ) * PI / 180;
                sn = sin(deg);
                cs = cos(deg);
                action_rotate(gpu_image_input, gpu_image_output, width, height,sn,cs);
                rotate_angle_l = rotate_angle;
            }*/
            if (r_l - r != 0 || g_l - g != 0 || b_l - b != 0) {
                applyKernelEqualizar(gpu_image_input, gpu_image_output, width, height, r_l - r, g_l - g, b_l - b);
                r_l = r;
                g_l = g;
                b_l = b;
            }
            if(scale_l - scale != 0){
                int width_zoom = width * scale;
                int height_zoom = height * scale;
                image_scale = Mat(height_zoom, width_zoom, CV_8UC3, Scalar(0, 0, 0));
                unsigned char *gpu_imgScale;
                unsigned char *cpu_imgScale = (unsigned char*)malloc(sizeof(unsigned char) * height_zoom * width_zoom * 3);
                hipMalloc( (void**)&gpu_imgScale, width_zoom * height_zoom * 3 * sizeof(unsigned char));
                
                dim3 blockDimsScale(512, 1, 1);
                dim3 gridDimsScale((unsigned int) ceil((double)(width_zoom * height_zoom * 3 / blockDims.x)), 1, 1 );
                kernel_resize <<< gridDimsScale, blockDimsScale>>>(gpu_image_input, gpu_imgScale, width, height, width_zoom, height_zoom);
                hipMemcpy( cpu_imgScale, gpu_imgScale, width_zoom * height_zoom * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );
                loadMatFromArray(image_scale, cpu_imgScale, width_zoom, height_zoom);
                hipFree( gpu_imgScale);
                imshow( "Scale", image_scale );
                scale_l = scale;
            }
        }
        image_canvas = cv::Mat(CANVAS_WIDTH, CANVAS_HEIGHT, CV_8UC3, Scalar(40, 40, 40));
        float factor;
        int desp_x, desp_y;
        if (width < height) {
            factor = CANVAS_HEIGHT * 1.0 / height;
            desp_x = (CANVAS_WIDTH - width * factor ) / 2;
            desp_y = 0;
        } else {
            factor = CANVAS_WIDTH  * 1.0 / width;
            desp_x = 0;
            desp_y = (CANVAS_HEIGHT - height * factor) / 2;
        }
        cv::resize(image_output, image_temp, Size(width * factor, height * factor)); //, interpolation = cv.INTER_CUBIC);
        image_temp.copyTo(image_canvas(cv::Rect(desp_x, desp_y, image_temp.cols, image_temp.rows)));
        cvui::image(frame, COLUMN_2, 10, image_canvas);
        image_histogram =  calcHist(image_output);

        //cvui::text(frame, COLUMN_1, 780, message,0.5,theColor);
        cvui::printf(frame, COLUMN_1, 780, "Finished in(ms)  = %.2f", milliseconds);
        cvui::window(frame, COLUMN_1 / 2, 800, COLUMN_2 - 10, 170, "Histograma");
        cvui::image(frame, COLUMN_1, 820, image_histogram);

        cvui::update();
        cv::imshow(WINDOW_NAME, frame);
        tecla = waitKey(50);
        if (tecla == 27) {
            break;
        }
    }
    return 0;
}